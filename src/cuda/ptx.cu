#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <bitset>
#include <string>

#include "encode_ptx.h"

#define NUM_THREADS 256
#define NUM_BLOCKS 1024


extern "C" bool is_cuda_available()
{
    hipDeviceProp_t prop;
    return hipGetDeviceProperties(&prop, 0) == hipSuccess;
}

extern "C" int sloth256_189_cuda_batch_encode(unsigned int piece[], size_t len,
                             const unsigned int iv[32], size_t layers)
{
    // for handling potential CUDA errors
    int cudaStatus;

    size_t block_count, thread_count;
    // there is `size` in the variable name, since len does not represent
    // the piece count, but instead the size of the piece_array (in bytes)
    size_t remaining_piece_size = len;
    // in bytes
    size_t processed_piece_size = 0;

    // 1 thread is responsible from 1 piece,
    // 1 thread handles 4096 bytes
    // 256 threads handle 1048576 bytes, or 2**20 bytes
    thread_count = NUM_THREADS;

    // 8GB as Bytes
    // allocating more than 8GB would be overkill, this is an upper-limit set for high-end GPUs.
    // we will tweak this down below with respect to the current available device.
    unsigned long long round_size = len;

    size_t free_mem, total_mem;

    // Getting free and total memory of the device
    if (hipMemGetInfo(&free_mem, &total_mem) != hipSuccess) {
        return 1;
    }

    //printf("\nFree memory on this device is: %llu Bytes\n", free_mem);
    //printf("Total memory on this device is: %llu Bytes\n", total_mem);  // we are not using this, but it is fancy :)

    // if device does not have enough free memory
    while (round_size > free_mem) {
        // make the memory requirement smaller
        round_size /= 2;
    }

    // Unfortunately, hipMalloc does not return an error when size is 0
    if (round_size == 0) {
        return 2;
    }
    //printf("Picked the default amount of memory to be allocated in each round as: %llu Bytes\n", round_size);

    // pointers for device
    u256 *d_piece = 0;
    u256 *d_iv = 0;

    // We want to keep thread_count at 256 for CUDA reasons so we are manipulating block_count instead.
    // (round_size >> 12) -> (round_size / 4096) -> piece_count
    // (piece_count / thread_count) -> how many blocks there should be
    block_count = (round_size / 4096) / thread_count;

    //printf("Trying to allocate %llu Bytes\n", round_size);
    // This might fail, due to User may have opened a program that heavily utilizes the GPU
    cudaStatus = hipMalloc(&d_piece, round_size);

    // If fails, reduce the requirement
    while (cudaStatus != hipSuccess) {
        cudaStatus = hipMalloc(&d_piece, round_size);
        round_size /= 2;
        block_count /= 2;
    }

    // IV occupies 32 bytes, piece occupies 4096 bytes.
    cudaStatus = hipMalloc(&d_iv, (round_size / 4096 * 32 ));

    if (cudaStatus != hipSuccess) {
        return 3;
    }

    while (true) {
        // Computing the next range of pieces to be processed, and copying them into GPU memory
        //
        // The reason for the extra division by 4 is: we are doing pointer arithmetic here. Type of `piece` and `iv`
        // are unsigned int, and unsigned int is allocating 4 bytes. So actually, iv+1 reaches to next unsigned int,
        // which is 4 bytes later and we have computed the actual size. We have to divide our computations by 4 in here.
        cudaStatus = hipMemcpy(d_piece, (piece + (processed_piece_size / 4)),
                                round_size, hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            cudaStatus = 4;
            break;
        }
        cudaStatus = hipMemcpy(d_iv, (iv + (processed_piece_size / 512)),
                                (round_size / 128), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            cudaStatus = 5;
            break;
        }

         // Calling the kernel, we cast (unsigned int) to suppress warning of possible data loss
        sloth256_189_encode_cuda<<<(unsigned int)block_count, (unsigned int)thread_count>>>(d_piece, d_iv, layers);

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            cudaStatus = 6;
            break;
        }

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            cudaStatus = 7;
            break;
        }

        cudaStatus = hipMemcpy((piece + (processed_piece_size / 4)), d_piece,
                                round_size, hipMemcpyDeviceToHost);
        // Copy back the result to host, again extra division by 4 because of pointer arithmetic
        if (cudaStatus != hipSuccess) {
            cudaStatus = 8;
            break;
        }

        processed_piece_size += round_size;
        remaining_piece_size -= round_size;

        if (remaining_piece_size == 0) {
            break;
        }
    }

    // cudaStatus is 0 if there is no error and other numbers for specific errors that we inspected for
    return cudaStatus;
}
