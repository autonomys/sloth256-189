#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <bitset>
#include <string>

#include "encode_ptx.h"

#define NUM_THREADS 256
#define NUM_BLOCKS 1024
#define EIGHT_GB_IN_BYTES 8589934592


extern "C" bool is_cuda_available()
{
    hipDeviceProp_t prop;
    return hipGetDeviceProperties(&prop, 0) == hipSuccess;
}

extern "C" int sloth256_189_cuda_batch_encode(unsigned int piece[], size_t len,
                             const unsigned int iv[32], size_t layers)
{   // len also represents how many bytes in piece[]

    int cudaStatus;  // for handling potential CUDA errors

    int block_count, thread_count;
    unsigned remaining_piece_size = len;  // there is `size` in the variable name, since len does not represent
    // the piece count, but instead the size of the piece_array (in bytes)
    unsigned processed_piece_size = 0;  // in bytes

    thread_count = NUM_THREADS;  // 1 thread is responsible from 1 piece,
    // 1 thread handles 4096 bytes
    // 256 threads handle 1048576 bytes, or 2**20 bytes

    unsigned long long default_round_size = EIGHT_GB_IN_BYTES;  // 8GB as Bytes
    // allocating more than 8GB would be overkill, this is an upper-limit set for high-end GPUs.
    // we will tweak this down below with respect to the current available device.

    unsigned long long to_be_processed_size, free_mem, total_mem;

    // Getting free and total memory of the device
    if (hipMemGetInfo(&free_mem, &total_mem) != hipSuccess) {
      return 1;
    }

    //printf("\nFree memory on this device is: %llu Bytes\n", free_mem);
    //printf("Total memory on this device is: %llu Bytes\n", total_mem);  // we are not using this, but it is fancy :)

    while (default_round_size > free_mem) {  // if device does not have enough free memory
        default_round_size /= 2;  // make the memory requirement smaller
    }

    //printf("Picked the default amount of memory to be allocated in each round as: %llu Bytes\n", default_round_size);

    block_count = (default_round_size / 4096) / thread_count;  // we want to keep thread_count at 256 for CUDA reasons
    // so we are manipulating block_count instead.
    // (to_be_processed_size >> 12) -> (to_be_processed_size / 4096) -> piece_count
    // (piece_count / thread_count) -> how many blocks there should be

    u256 *d_piece, *d_iv;  // pointers for device

    while (true) {  // don't panic, this is not an endless loop :)
        to_be_processed_size = default_round_size;  // at the start of the each turn, use the default size

        // it could be that, remaining_piece_size could be less than the default size
        if (remaining_piece_size < (to_be_processed_size)) {  // so we will adjust our worker count accordingly

            block_count = (remaining_piece_size / 4096) / thread_count;
            // since each thread will handle 4096 bytes, the above equation should make sense
            // important note in here: the above division should not produce a remainder
            // `thread_count` will be 256. During load balancing, send multiples of
            // 256 pieces to the GPU to be safe, so that the above division will not have any remainder

            to_be_processed_size = block_count * thread_count * 4096;  // update our variable
        }

        //printf("Trying to allocate %llu Bytes\n", to_be_processed_size);
        cudaStatus = hipMalloc(&d_piece, to_be_processed_size);  // trying to allocate memory
        // this might fail, due to User may have opened a program that heavily utilizes the GPU

        while ((cudaStatus != hipSuccess) && to_be_processed_size != 0) {  // if fails, reduce the requirement
            cudaStatus = hipMalloc(&d_piece, to_be_processed_size);
            to_be_processed_size /= 2;
            block_count /= 2;
        }

        if (to_be_processed_size == 0) {  // unfortunately, hipMalloc does not return an error when size is 0
            cudaStatus = 2;  // hipMalloc failed!
            break;
        }

        cudaStatus = hipMalloc(&d_iv, (to_be_processed_size / 128 ));
        // iv occupies 32 bytes, piece occupies 4096 bytes.
        // Instead of dividing the size into 4096, then multiplying it with 32, we can divide into 128.
        if (cudaStatus != hipSuccess) {
            cudaStatus = 3;  // hipMalloc failed!
            break;
        }

        // computing the next range of pieces to be processed, and copying them into GPU memory
        cudaStatus = hipMemcpy(d_piece, (piece + (processed_piece_size / 4)),
                                to_be_processed_size, hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            cudaStatus = 4;  // hipMemcpy failed!
            break;
        }
        cudaStatus = hipMemcpy(d_iv, (iv + (processed_piece_size / 512)),
                                (to_be_processed_size / 128), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            cudaStatus = 5;  // hipMemcpy failed!
            break;
        }
        // the reason for the extra division by 4 is:
        // we are doing pointer arithmetic here. Type of `piece` and `iv` are unsigned int, and unsigned int
        // is allocating 4 bytes. So actually, iv+1 reaches to next unsigned int, which is 4 bytes later
        // and we have computed the actual size. We have to divide our computations by 4 in here

        sloth256_189_encode_cuda<<<block_count, thread_count>>>(d_piece, d_iv);  // calling the kernel

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            // Kernel launch failed
            cudaStatus = 6;
            break;
        }

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            // hipDeviceSynchronize returned error code %d, CUDA operation did not finish correctly,
            // returning back to CPU...
            cudaStatus = 7;
            break;
        }

        cudaStatus = hipMemcpy((piece + (processed_piece_size / 4)), d_piece,
                                to_be_processed_size, hipMemcpyDeviceToHost);
        // copy back the result to host, again extra division by 4 because of pointer arithmetic
        if (cudaStatus != hipSuccess) {
            cudaStatus = 8;  // hipMemcpy failed!
            break;
        }

        processed_piece_size += to_be_processed_size;  // update the processed_piece_size
        remaining_piece_size -= to_be_processed_size;  // likewise :)

        if (remaining_piece_size == 0) {  // successful!
            break;  // Hurry! Get out of the loop
        }
    }

    hipFree(d_piece);  // clean-up
    hipFree(d_iv);  // clean-up

    return cudaStatus;  // cudaStatus is 0 if there is no error, 1 if there is error
    // and other numbers for specific errors that we inspected for
}
