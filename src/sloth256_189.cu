#include <hip/hip_runtime.h>

extern "C" bool detect_cuda()
{
    hipDeviceProp_t prop;
    return hipGetDeviceProperties(&prop, 0) == hipSuccess;
}
