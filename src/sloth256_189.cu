#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "sloth256_189.c"

__global__ void test_1x1_kernel(unsigned char *piece_n_iv, size_t len,
                                size_t layers)
{
    (void)sloth256_189_encode(piece_n_iv, len, piece_n_iv+len, layers);
}

extern "C" bool test_1x1_cuda(unsigned char piece[], size_t len,
                              const unsigned char iv[32], size_t layers)
{
    unsigned char *piece_n_iv;

    if (hipMalloc(&piece_n_iv, len+32) != hipSuccess)
        return false;

    hipMemcpy(piece_n_iv, piece, len, hipMemcpyHostToDevice);
    hipMemcpy(piece_n_iv+len, iv, 32, hipMemcpyHostToDevice);

    test_1x1_kernel<<<1, 1>>>(piece_n_iv, len, layers);

    if (hipDeviceSynchronize() == hipSuccess)
        hipMemcpy(piece, piece_n_iv, len, hipMemcpyDeviceToHost);

    hipFree(piece_n_iv);

    return true;
}

extern "C" bool detect_cuda()
{
    hipDeviceProp_t prop;
    return hipGetDeviceProperties(&prop, 0) == hipSuccess;
}
