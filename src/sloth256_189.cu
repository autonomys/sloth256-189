#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "sloth256_189.c"

__global__ void test_1x1_kernel(unsigned char *piece_n_iv, size_t len,
                                size_t layers)
{
    (void)sloth256_189_encode(piece_n_iv, len, piece_n_iv+len, layers);
}

extern "C" void test_1x1_cuda(unsigned char piece[], size_t len,
                              const unsigned char iv[32], size_t layers)
{
    unsigned char *piece_n_iv;

    hipMalloc(&piece_n_iv, len+32);
    hipMemcpy(piece_n_iv, piece, len, hipMemcpyHostToDevice);
    hipMemcpy(piece_n_iv+len, iv, 32, hipMemcpyHostToDevice);

    test_1x1_kernel<<<1, 1>>>(piece_n_iv, len, layers);

    hipDeviceSynchronize();

    hipMemcpy(piece, piece_n_iv, len, hipMemcpyDeviceToHost);
}

extern "C" bool detect_cuda()
{
    hipDeviceProp_t prop;
    return hipGetDeviceProperties(&prop, 0) == hipSuccess;
}
